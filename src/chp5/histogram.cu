
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/* Each thread writes to one block of 256 elements of global memory and contends for write access */

__global__ void myhistogram256kernel_01(const unsigned char const * d_hist_data,
		unsigned int * const d_bin_data) {
	/* Work out our thread id */
	const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	const unsigned int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
	const unsigned int tid = idx + idy * blockDim.x * gridDim.x;

	/* Fetch the data value */
	const unsigned char value = d_hist_data[tid];
	atomicAdd(&(d_bin_data[value]), 1);
}

/* Each read is 4 bytes, not one, 32 x 4 = 128 byte reads */
__global__ void myhistogram256kernel_02(const unsigned char const * d_hist_data,
		unsigned int * const d_bin_data) {
	/* Work out our thread id */
	const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	const unsigned int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
	const unsigned int tid = idx + idy * blockDim.x * gridDim.x;

	/* Fetch the data value */
	const unsigned int value_u32 = d_hist_data[tid];
	atomicAdd(&(d_bin_data[((value_u32 & 0x000000FF))]), 1);
	atomicAdd(&(d_bin_data[((value_u32 & 0x0000FF00 >> 8))]), 1);
	atomicAdd(&(d_bin_data[((value_u32 & 0x00FF0000 >> 16))]), 1);
	atomicAdd(&(d_bin_data[((value_u32 & 0xFF000000 >> 24))]), 1);
}

__shared__ unsigned int d_bin_data_shared[256];

/* Each read is 4 bytes, not one, 32 x 4 = 128 byte reads */
__global__ void myhistogram256kernel_03(const unsigned char const * d_hist_data,
		unsigned int * const d_bin_data) {
	/* Work out our thread id */
	const unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	const unsigned int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
	const unsigned int tid = idx + idy * blockDim.x * gridDim.x;

	/* Clear shared memory */
	d_bin_data_shared[threadIdx.x] = 0;

	/* Fetch the data value */
	const unsigned int value_u32 = d_hist_data[tid];

	/* Wait for all threads to update shared memory */
	__syncthreads();

	atomicAdd(&(d_bin_data[((value_u32 & 0x000000FF))]), 1);
	atomicAdd(&(d_bin_data[((value_u32 & 0x0000FF00 >> 8))]), 1);
	atomicAdd(&(d_bin_data[((value_u32 & 0x00FF0000 >> 16))]), 1);
	atomicAdd(&(d_bin_data[((value_u32 & 0xFF000000 >> 24))]), 1);

	/* Wait for all threads to update shared memory */
	__syncthreads();

	/* The write the accumulated data back to global memory in blocks, not scattered */
	atomicAdd(&(d_bin_data[threadIdx.x]), d_bin_data_shared[threadIdx.x]);
}

/* Each read is 4 bytes, not one, 32 x 4 = 128 byte reads */
__global__ void myhistogram256kernel_07(const unsigned int const * d_hist_data,
		unsigned int * const d_bin_data, unsigned int N) {
	/* Work out our thread id */
	const unsigned int idx = (blockIdx.x * (blockDim.x * N)) + threadIdx.x;
	const unsigned int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
	const unsigned int tid = idx + idy * (blockDim.x * N) * gridDim.x;

	/* Clear shared memory */
	d_bin_data_shared[threadIdx.x] = 0;

	/* Wait for all threads to update shared memory */
	__syncthreads();

	for (unsigned int i = 0, tid_offset = 0; i < N; i++, tid_offset += 256) {
		const unsigned int value_u32 = d_hist_data[tid + tid_offset];
		atomicAdd(&(d_bin_data[((value_u32 & 0x000000FF))]), 1);
		atomicAdd(&(d_bin_data[((value_u32 & 0x0000FF00 >> 8))]), 1);
		atomicAdd(&(d_bin_data[((value_u32 & 0x00FF0000 >> 16))]), 1);
		atomicAdd(&(d_bin_data[((value_u32 & 0xFF000000 >> 24))]), 1);
	}

	/* Wait for all threads to update shared memory */
	__syncthreads();

	/* The write the accumulated data back to global memory in blocks, not scattered */
	atomicAdd(&(d_bin_data[threadIdx.x]), d_bin_data_shared[threadIdx.x]);
}

int main() {

	return EXIT_SUCCESS;
}

