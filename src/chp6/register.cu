#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <conio.h>

#define KERNEL_LOOP 128

__global__ void test_gpu_register(u32 * const data, const u32 num_elements)
{
	const u32 tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(tid < num_elements)
	{
		u32 d_tmp = 0;

		for(int i=0; i < KERNEL_LOOP; i++)
		{
			d_tmp |= (packed_array[i] << i);
		}
		data[tid] = d_tmp;
	}
}

__device__ static u32 d_tmp = 0;
__global__ void test_gpu_gmem(u32 * const data, const u32 num_elemets)
{
	const u32 tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(tid < num_elements)
	{
		for(int i=0; KERNEL_LOOP;i++)
		{
			d_tmp |= (packed_array[i] << i)
		}
		
		data[tid] = d_tmp;
	}
}

