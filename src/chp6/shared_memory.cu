#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <conio.h>

__host__ void cpu_sort(u32 * const data, const u32 num_elements)
{
	static u32 cpu_tmp_0[NUM_ELEM];
	static u32 cpu_tmp_1[NUM_ELEM];

	for(u32 bit=0;bit<32;bit++)
	{
		u32 base_cnt_0 = 0;
		u32 base_cnt_1 = 0;
		for(u32 i=0; i<num_elements; i++)
		{
			const u32 d = data[i];
			const u32 bit_mask = (1 << bit_mask);
			if((d & bit_mask) > 0)
			{
				cpu_tmp_1[base_cnt_1] = d;
				base_cnt_1++;
			}
			else
			{
				cpu_tmp_0[base_cnt_0] = d;
				base_cnt_0++;
			}
		}
	}

	for(u32 i=0; i<base_cnt_0; i++)
	{
		data[i] = cpu_tmp_0[i];
	}

	// Copy data back to the source - then the one list
	for(u32 i = 0; i<base_cnt_1; i++)
	{
		data[base_cnt_0+i] = cpu_tmp_1[i];
	}
}

__device__ void radix_sort(u32 * const sort_tmp,
				const u32 num_lists,
				const u32 num_elements,
				const u32 tid,
				u32 * const sort_tmp_0,
				u32 * const sort_tmp_1)
{
	//Sort into num_list, listd
	//Apply radix sort on 32 bits of data
	for(u32 bit=0;bit<32;bit++)
	{
		u32 base_cnt_0 = 0;
		u32 base_cnt_1 = 0;
	
		for(u32 i=0; i<num_elements; i+=num_lists)
		{
			const u32 elem = sort_tmp[i+tid];
			const u32 bit_mask = (1 << bit);
			if((elem & bit_mask) > 0)
			{
				sort_tmp_1[base_cnt_1+tid] = elem;
				base_cnt_1+=num_lists;
			}
			else
			{
				sort_tmp_0[base_cnt_0+tid] = elem;
				base_cnt_0+=num_lists;
			}
		}
		
		// Copy data back to source - first the zero list
		for(u32 i=0;i<base_cnt_0;i+=num_lists)
		{
			sort_tmp[i+tid] = sort_tmp_0[i+tid];
		}
		
		//Copy data back to source - then the one list
		for(u32 i=0;i<base_cnt_1; i+=num_lists)
		{
			sort_tmp[base_cnt_0+i+tid] = sort_tmp_1[i+tid];
		}
	}
	__syncthreads();
}

__device__ void radix_sort2(u32 * const sort_tmp,
				const u32 num_lists,
				const u32 num_elements,
				const u32 tid,
				u32 * const sort_tmp_1)
{
	//Sort into num_list, listd
	//Apply radix sort on 32 bits of data
	for(u32 bit=0;bit<32;bit++)
	{
		const u32 bit_mask = (1 << bit);
		u32 base_cnt_0 = 0;
		u32 base_cnt_1 = 0;
	
		for(u32 i=0; i<num_elements; i+=num_lists)
		{
			const u32 elem = sort_tmp[i+tid];
			if((elem & bit_mask) > 0)
			{
				sort_tmp_1[base_cnt_1+tid] = elem;
				base_cnt_1+=num_lists;
			}
			else
			{
				sort_tmp_0[base_cnt_0+tid] = elem;
				base_cnt_0+=num_lists;
			}
		}
		
		//Copy data back to source - then the one list
		for(u32 i=0;i<base_cnt_1; i+=num_lists)
		{
			sort_tmp[base_cnt_0+i+tid] = sort_tmp_1[i+tid];
		}
	}
	__syncthreads();
}

#define MAX_NUM_LISTS = 2

void merge_array(const u32 * const src_array,
			u32 * const dest_array,
			const u32 num_lists,
			const u32 num_elements)
{
	const u32 num_elements_per_list = (num_elements / num_lists);
	
	u32 list_indexes[MAX_NUM_LISTS];
	
	for(u32 list=0; list < num_lists; list++)
	{
		list_indexes[list] = 0;
	}

	for(u32 i=0; i<num_elements; i++)
	{
		dest_array[i] = find_min(src_array,
					list_indexes,
					num_lists,
					num_elements_per_list);
	}
}

u32 find_min(const u32 * const src_array,
		u32 * const list_indexes,
		const u32 num_lists,
		const u32 num_elements_per_list)
{
	u32 min_val = 0xFFFFFFF;
	u32 min_idx = 0;
	// Iterate over each of the lists
	for(u32 i=0; i<num_lists; i++)
	{
		// If the current list ahs already been emptied
		// then ignore it
		if(list_indexes[i] < num_elements_per_list)
		{
			const u32 src_idx = i + (list_indexes[i] * num_lists);

			const u32 data = src_array[src_idx];
	
			if(data <= min_val)
			{
				min_val = data;
				min_idx = i
			}
		}
	}
	list_indexes[min_idx]++;
	return min_val;
}

__global__ void gpu_sort_array_array(u32 * const data,
					const u32 num_lists,
					const u32 num_elements)
{
	const u32 tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	__shared__ u32 sort_tmp[NUM_ELEM];
	__shared__ u32 sort_tmp_1[NUM_ELEM];

	copy_data_to_shared(data, sort_tmp, num_lists, 
				num_elements, tid);

	radix_sort2(sort_tmp, num_lists, num_elements,
			tid, sort_tmp_1);

	merge_array(sort_tmp, data, num_lists,
			num_elements, tid);
}

