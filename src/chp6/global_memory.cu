#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>

static const int WORK_SIZE = 256;

#define NUM_ELEMENTS 4096

typedef struct {
	u32 a;
	u32 b;
	u32 c;
	u32 d;
} INTERLEAVED_T;

typedef INTERLEAVED_T INTERLEAVED_ARRAY_T[NUM_ELEMENTS];

typedef u32 ARRAY_MEMBER_T[NUM_ELEMENTS];

typedef struct {
	ARRAY_MEMBER_T a;
	ARRAY_MEMBER_T b;
	ARRAY_MEMBER_T c;
	ARRAY_MEMBER_T d;
} NON_INTERLEAVED_T;

__host__ float add_test_non_interleaved_cpu(
		NON_INTERLEAVED_T * const host_dest_ptr,
		const NON_INTERLEAVED_T * const host_src_ptr, const u32 iter,
		const u32 num_elements) {
	float start_time = get_time();

	for (u32 tid = 0; tid < num_elements; tid++) {
		for (u32 i = 0; i < iter; i++) {
			host_dest_ptr->a[tid] += host_src_ptr->a[tid];
			host_dest_ptr->b[tid] += host_src_ptr->b[tid];
			host_dest_ptr->c[tid] += host_src_ptr->c[tid];
			host_dest_ptr->d[tid] += host_src_ptr->d[tid];
		}
	}

	const float delta = get_time() - start_time;

	return delta;
}

__host__ float add_test_interleaved_cpu(INTERLEAVED_T * const host_dest_ptr,
		const INTERLEAVED_T * const host_src_ptr, const u32 iter,
		const u32 num_elements) {
	float start_time = get_time();

	for (u32 tid = 0; tid < num_elements; tid++) {
		for (u32 i = 0; i < iter; i++) {
			host_dest_ptr[tid].a += host_src_ptr[tid].a;
			host_dest_ptr[tid].b += host_src_ptr[tid].b;
			host_dest_ptr[tid].c += host_src_ptr[tid].c;
			host_dest_ptr[tid].d += host_src_ptr[tid].d;
		}
	}

	const float delta = get_time() - start_time;

	return delta;
}

__global__ void add_kernel_interleaved(INTERLEAVED_T * const dest_ptr,
		const INTERLEAVED_T * const src_ptr, const u32 iter,
		const u32 num_elements) {
	float start_time = get_time();

	const u32 tid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(tid < num_elements)
	{
		for(u32 i=0; i<iter; i++)
		{
			dest_ptr[tid].a += src_ptr[tid].a;
			dest_ptr[tid].b += src_ptr[tid].b;
			dest_ptr[tid].c += src_ptr[tid].c;
			dest_ptr[tid].d += src_ptr[tid].d;
		}
	}
}

__global__ void add_kernel_non_interleaved(
		NON_INTERLEAVED_T * const dest_ptr,
		const NON_INTERLEAVED_T * const src_ptr, const u32 iter,
		const u32 num_elements) {
	float start_time = get_time();

	for (u32 tid = 0; tid < num_elements; tid++) {
		for (u32 i = 0; i < iter; i++) {
			dest_ptr->a[tid] += src_ptr->a[tid];
			dest_ptr->b[tid] += src_ptr->b[tid];
			dest_ptr->c[tid] += src_ptr->c[tid];
			dest_ptr->d[tid] += src_ptr->d[tid];
		}
	}
}

__host__ float add_test_interleaved(INTERLEAVED_T * const host_dest_ptr,
		const INTERLEAVED_T * const host_src_ptr, const u32 iter,
		const u32 num_elements)
{
	const u32 num_threads = 256;
	const u32 num_blocks = (num_elements + (num_threads-1)) / num_threads;

	const size_t num_bytes = (sizeof(INTERLEAVED_T) * num_elements);
	INTERLEAVED_T * device_dest_ptr;
	INTERLEAVED_T * device_src_ptr;

	CUDA_CALL(hipMalloc((void **) &device_src_ptr, num_bytes));
	CUDA_CALL(hipMalloc((void **) &device_dest_ptr, num_bytes));

	hipEvent_t kernel_start, kernel_stop;
	hipEventCreate(&kernel_start1,0);
	hipEventCreate(&kernel_start2,0);

	hipStream_t test_stream;
	CUDA_CALL(hipStreamCreate(&test_stream));

	CUDA_CALL(hipMemcpy(device_src_ptr, host_src_ptr, num_bytes,hipMemcpyHostToDevice));

	CUDA_CALL(hipEventRecord(kernel_start, 0));

	add_kernel_interleaved<<<num_blocks,num_threads>>>(device_dest_ptr, device_src_ptr, iter, num_elements);

	CUDA_CALL(hipEventRecord(kernel_stop, 0));

	CUDA_CALL(hipEventSynchronize(kernel_stop));

	float delta = 0.0F;
	CUDA_CALL(hipEventElapsedTime(&delta, kernel_start, kernel_stop));

	CUDA_CALL(hipFree(device_src_ptr));
	CUDA_CALL(hipFree(device_dest_ptr));
	CUDA_CALL(hipEventDestroy(kernel_start));
	CUDA_CALL(hipEventDestroy(kernel_stop));
	CUDA_CALL(hipStreamDestroy(test_stream));

	return delta;
}


/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CALL(x) {														\
	hipError_t _m_cudaStat = x;											\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

__host__ __device__ unsigned int bitreverse(unsigned int number) {
	number = ((0xf0f0f0f0 & number) >> 4) | ((0x0f0f0f0f & number) << 4);
	number = ((0xcccccccc & number) >> 2) | ((0x33333333 & number) << 2);
	number = ((0xaaaaaaaa & number) >> 1) | ((0x55555555 & number) << 1);
	return number;
}

/**
 * CUDA kernel function that reverses the order of bits in each element of the array.
 */
__global__ void bitreverse(void *data) {
	unsigned int *idata = (unsigned int*) data;
	idata[threadIdx.x] = bitreverse(idata[threadIdx.x]);
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(void) {
	void *d = NULL;
	int i;
	unsigned int idata[WORK_SIZE], odata[WORK_SIZE];

	for (i = 0; i < WORK_SIZE; i++)
		idata[i] = (unsigned int) i;

	CUDA_CHECK_RETURN(hipMalloc((void** ) &d, sizeof(int) * WORK_SIZE));
	CUDA_CHECK_RETURN(
			hipMemcpy(d, idata, sizeof(int) * WORK_SIZE,
					hipMemcpyHostToDevice));

	bitreverse<<<1, WORK_SIZE, WORK_SIZE * sizeof(int)>>>(d);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());
	CUDA_CHECK_RETURN(
			hipMemcpy(odata, d, sizeof(int) * WORK_SIZE,
					hipMemcpyDeviceToHost));

	for (i = 0; i < WORK_SIZE; i++)
		printf("Input value: %u, device output: %u, host output: %u\n",
				idata[i], odata[i], bitreverse(idata[i]));

	CUDA_CHECK_RETURN(hipFree((void* ) d));
	CUDA_CHECK_RETURN(hipDeviceReset());

	return 0;
}
