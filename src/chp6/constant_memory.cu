#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include "const_common.h"
#include "conio.h"
#include "assert.h"

static const int WORK_SIZE = 256;

#define CUDA_CALL(x) 														\
{																			\
	hipError_t _m_cudaStat = x;											\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }																		\

#define KERNEL_LOOP 65536

__constant__ static const u32 const_data_01 = 0x55555555;
__constant__ static const u32 const_data_02 = 0x77777777;
__constant__ static const u32 const_data_03 = 0x33333333;
__constant__ static const u32 const_data_04 = 0x11111111;


__global__ void const_test_gpu_literal(u32 * const data, const u32 num_elements)
{
	const u32 tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(tid < num_elements)
	{
		u32 d = 0x55555555;

		for(int i=0;i<KERNEL_LOOP;i++)
		{
			d ^= 0x55555555;
			d |= 0x77777777;
			d &= 0x33333333;
			d |= 0x11111111;
		}

		data[tid] = d;
	}
}

__global__ void const_test_gpu_const(u32 * const data, const u32 num_elements)
{
	const u32 tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(tid < num_elements)
	{
		u32 d = const_data_01;

		for(int i=0;i<KERNEL_LOOP;i++)
		{
			d ^= const_data_01;
			d |= const_data_02;
			d &= const_data_03;
			d |= const_data_04;
		}

		data[tid] = d;
	}
}

__host__ void gpu_kernel(void)
{
	const u32 num_elements = (128*1024);
	const u32 num_threads = 256;
	const u32 num_blocks = (num_elements + (num_threads-1))/num_threads;
	const u32 num_bytes = num_elements * sizeof(u32);
	int max_device_num;
	const int max_runs = 6;

	CUDA_CALL(hipGetDeviceCount(&max_device_num));

	for(int device_num=0; device_num < max_device_num; device_num++)
	{
		CUDA_CALL(hipSetDevice(device_num));

		for(int num_test=0; num_test < max_runs; num_test++)
		{
			u32 * data_gpu;
			hipEvent_t kernel_start1, kernel_stop1;
			hipEvent_t kernel_start2, kernel_stop2;
			float delta_time1 = 0.0f, delta_time2 = 0.0F;
			struct hipDeviceProp_t device_prop;
			char device_prefix[261];

			CUDA_CALL(hipMalloc(&data_gpu, num_bytes));
			CUDA_CALL(hipEventCreate(&kernel_start1));
			CUDA_CALL(hipEventCreate(&kernel_start2));
			CUDA_CALL(hipEventCreateWithFlags(&kernel_stop1, hipEventBlockingSync));
			CUDA_CALL(hipEventCreateWithFlags(&kernel_stop2, hipEventBlockingSync));

			CUDA_CALL(hipGetDeviceProperties(&device_prop, device_num));
			sprintf(device_prefix, "ID: %d %s:", device_num, device_prop.name);

			const_test_gpu_literal<<< num_blocks, num_threads >>>(data_gpu, num_elements);

			cuda_error_check("Error ", " returned from literal startup  kernel!");

			CUDA_CALL(hipEventRecord(kernel_start1,0));
			const_test_gpu_literal <<<num_blocks, num_threads>>>(data_gpu, num_elements);

			cuda_error_check("Error ", " returned from literal runtime  kernel!");

			CUDA_CALL(hipEventRecord(kernel_stop1,0));
			CUDA_CALL(hipEventSynchronize(kernel_stop1));
			CUDA_CALL(hipEventElapsedTime(&delta_time1, kernel_start1, kernel_stop1));

			const_test_gpu_const<<< num_blocks, num_threads >>>(data_gpu, num_elements);

			cuda_error_check("Error ", " returned from literal startup  kernel!");

			CUDA_CALL(hipEventRecord(kernel_stop2,0));
			CUDA_CALL(hipEventSynchronize(kernel_stop2));
			CUDA_CALL(hipEventElapsedTime(&delta_time2, kernel_start2, kernel_stop2));

			if(delta_time1 > delta_time2)
			{
				printf("\n%sConstant version is faster by: %.2fms (Const=%.2fms vs. Literal=%.2fms)",device_prefix, delta_time1-delta_time2, delta_time1, delta_time2);
			}
			else
			{
				printf("\n%sLiteral version is faster by: %.2fms (Const=%.2fms vs. Literal=%.2fms)",device_prefix, delta_time2-delta_time1, delta_time1, delta_time2);
			}

			CUDA_CALL(hipEventDestroy(kernel_start1));
			CUDA_CALL(hipEventDestroy(kernel_start2));
			CUDA_CALL(hipEventDestroy(kernel_stop1));
			CUDA_CALL(hipEventDestroy(kernel_stop2));
			CUDA_CALL(hipFree(data_gpu));
		}

		CUDA_CALL(hipDeviceReset());
		printf("\n");
	}
	wait_exit();
}

__device__ static u32 data_01 = 0x55555555;
__device__ static u32 data_02 = 0x77777777;
__device__ static u32 data_03 = 0x33333333;
__device__ static u32 data_04 = 0x11111111;

__global__ void const_test_gpu_gmem(u32 * const data, const u32 num_elements)
{
	const u32 tid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(tid < num_elements)
	{
		u32 d = data_01;

		for(int i=0;i<KERNEL_LOOP;i++)
		{
			d ^= data_01;
			d |= data_02;
			d &= data_03;
			d |= data_04;
		}

		data[tid] = d;
	}
}

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

__host__ __device__ unsigned int bitreverse(unsigned int number) {
	number = ((0xf0f0f0f0 & number) >> 4) | ((0x0f0f0f0f & number) << 4);
	number = ((0xcccccccc & number) >> 2) | ((0x33333333 & number) << 2);
	number = ((0xaaaaaaaa & number) >> 1) | ((0x55555555 & number) << 1);
	return number;
}

/**
 * CUDA kernel function that reverses the order of bits in each element of the array.
 */
__global__ void bitreverse(void *data) {
	unsigned int *idata = (unsigned int*) data;
	idata[threadIdx.x] = bitreverse(idata[threadIdx.x]);
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(void) {
	void *d = NULL;
	int i;
	unsigned int idata[WORK_SIZE], odata[WORK_SIZE];

	for (i = 0; i < WORK_SIZE; i++)
		idata[i] = (unsigned int) i;

	CUDA_CHECK_RETURN(hipMalloc((void**) &d, sizeof(int) * WORK_SIZE));
	CUDA_CHECK_RETURN(
			hipMemcpy(d, idata, sizeof(int) * WORK_SIZE, hipMemcpyHostToDevice));

	bitreverse<<<1, WORK_SIZE, WORK_SIZE * sizeof(int)>>>(d);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());
	CUDA_CHECK_RETURN(hipMemcpy(odata, d, sizeof(int) * WORK_SIZE, hipMemcpyDeviceToHost));

	for (i = 0; i < WORK_SIZE; i++)
		printf("Input value: %u, device output: %u, host output: %u\n",
				idata[i], odata[i], bitreverse(idata[i]));

	CUDA_CHECK_RETURN(hipFree((void*) d));
	CUDA_CHECK_RETURN(hipDeviceReset());

	return 0;
}
